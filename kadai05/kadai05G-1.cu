
#include <hip/hip_runtime.h>
#include<stdio.h>

// カーネル関数
__global__ void f(void){
    printf("Hello, World!\n");
}

int main(void){
    // カーネル関数の呼び出し
    f<<<2,4>>>();
    // カーネル関数の終了を待つ
    hipDeviceSynchronize();
    return 0;
}