
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>

#define N 8

//配列を表示する
void printArray(int a[],int size,const char *str){
    printf("%s",str);
    for(int i=0; i<size; i++){
        printf("%4d ", a[i]);
    }
    printf("\n");
}

//配列を初期化する関数
void initArray(int a[],int size){
    for(int i=0; i<size; i++){
        //0以上100未満の乱数を生成
        a[i] =(int)((rand()/(RAND_MAX+1.0))*100);
    }
}

//a[]+b[]の結果をc[]に格納する
__global__ void addArray(int *a ,int *b,int *c){
    //一意識別番号の取得
    int id=blockIdx.x * blockDim.x+threadIdx.x;

    if(id<N){
        c[id]=a[id]+b[id];
    }
}

int main(void){
    //ホスト上に確保した日あ列
    int a[N],b[N],c[N];

    //デバイス上に確保した配列へのポインタ
    int *adev,*bdev,*cdev;

    //デバイス上に配列領域を確保し，その領域の先頭アドレスを取得する
    hipMalloc(&adev, N * sizeof(int));
    hipMalloc(&bdev, N * sizeof(int));
    hipMalloc(&cdev, N * sizeof(int));

    //ホスト上の配列に初期値を設定する
    initArray(a, N);
    initArray(b, N);
    printArray(a,N,"a");
    printArray(b,N,"b");

    //a,bの内容をデバイスメモリ上の配列にコピーする
    hipMemcpy(adev, a, N * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(bdev, b, N * sizeof(int), hipMemcpyHostToDevice);

    //カーネル関数の変数呼び出し
    addArray<<<4,4>>>(adev,bdev,cdev);
    hipDeviceSynchronize();

    //計算結果(cdevが指す領域)をホスト側にコピーする
    hipMemcpy(c, cdev, N * sizeof(int), hipMemcpyDeviceToHost);
    printArray(c, N, "c");

    //デバイス上に確保した配列領域を解放する
    hipFree(adev);
    hipFree(bdev);
    hipFree(cdev);

    return 0;
}