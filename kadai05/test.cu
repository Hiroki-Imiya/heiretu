
#include <hip/hip_runtime.h>
#include<stdio.h>
#define N 10

//配列を表示する
void printArray(int *array, int size){
    for(int i=0; i<size; i++){
        printf("%d ", array[i]);
    }
    printf("\n");
}

//gpu上で実行する関数，__global__を付けるとgpu上で実行される関数を表す
__global__ void f(int k,int *p,int n){
    //自スレッドの識別番号を取得
    int myid = blockIdx.x * blockDim.x + threadIdx.x;

    //自分が担当する要素に処理を施す
    if(0<=myid && myid<n){
        p[myid] = p[myid] * k;
    }
}

int main(void){
    int i,a[N];

    //デバイス(gpu)上のメモリアドレス.配列aをgpu上にコピーする場合，
    //aのデバイス上の領域という意味でよくa_devという名前を使う
    int *a_dev;

    //配列a(cpu)の初期化
    for(i=0; i<N; i++){
        a[i] = i;
    }
    printf("before: ");printArray(a,N);

    //gpu上に整数10個分の領域を確保し，そのアドレスをa_devに代入
    hipMalloc(&a_dev, N*sizeof(int));

    //配列aをgpu上にコピー
    hipMemcpy(a_dev, a, N*sizeof(int), hipMemcpyHostToDevice);

    //関数fをgpu上で並列に呼び出しする．1×NのN並列で呼び出す
    f<<<1,N>>>(100,a_dev,N);

    //gpu上での処理が終わるまで待つ
    hipDeviceSynchronize();

    //gpu上の配列a_devをcpu上の配列aにコピー
    hipMemcpy(a, a_dev, N*sizeof(int), hipMemcpyDeviceToHost);

    printf("after: ");printArray(a,N);

    //gpu上で確保した領域を解放
    hipFree(a_dev);

    return 0;
}
