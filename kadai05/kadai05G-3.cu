
#include <hip/hip_runtime.h>
#include<stdio.h>

// カーネル関数
__global__ void f(void){
    //一意識別番号を取得
    int myid = blockIdx.x * blockDim.x + threadIdx.x;

    //一意識別番号　その計算に用いた値を表示
    printf("myid = %-2d , bDim=(%d %d %d) , bIdx=(%d %d %d) , tIdx=(%d %d %d)\n",
        myid,
        blockDim.x, blockDim.y, blockDim.z,
        blockIdx.x, blockIdx.y, blockIdx.z,
        threadIdx.x, threadIdx.y, threadIdx.z
    );
}

int main(void){
    // カーネル関数の呼び出し 3ブロック4スレッド
    f<<<3,4>>>();
    // カーネル関数の終了を待つ
    hipDeviceSynchronize();
    return 0;
}