
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>

//行列のサイズ
#define N 16

//x方向のブロック内のスレッド数
#define BSX 2
//y方向のブロック内のスレッド数
#define BSY 2

//行列を表示する(N<=16のときのみ)
void printMatrix(float a[][N],const char *str){
#if (N<=16)
    printf("==== %s ====\n",str);
    for(int i=0; i<N; i++){
        for(int j=0; j<N; j++){
            printf("%4.0f ", a[i][j]);
        }
        printf("\n");
    }
#endif
}

//行列を初期化する関数
void initMatrix(float a[][N]){
    for(int i=0; i<N; i++){
        for(int j=0; j<N; j++){
            //0以上10未満の乱数を生成
            a[i][j] =(float)((rand()/(RAND_MAX+1.0))*10);
        }
    }
}

//行列積を計算するカーネル関数
__global__ void mulMatrix(float (*a)[N],float (*b)[N],float (*c)[N]){
    
    int xid=blockIdx.x * blockDim.x+threadIdx.x;
    int yid=blockIdx.y * blockDim.y+threadIdx.y;

    //担当部分の行列積を計算
    if(xid < N && yid < N){
        c[yid][xid]=0.0;
        for(int k=0;k<N;k++){
            c[yid][xid] += a[yid][k] * b[k][xid];
        }
    }
}

int main(void){
    //ホスト上に確保した2次元配列（行列）
    float a[N][N],b[N][N],c[N][N],cs[N][N];

    float (*adev)[N], (*bdev)[N], (*cdev)[N]; //デバイス上に確保した2次元配列を指すポインタ

    //ホスト上の配列に初期値を設定する
    initMatrix(a);
    initMatrix(b);
    printMatrix(a, "a");
    printMatrix(b, "b");

    ////// ここからGPU版行列積 ///////////////////////

    //デバイス上に配列領域を確保し，その領域の先頭アドレスを取得する
    hipMalloc(&adev, sizeof(float)*N*N);
    hipMalloc(&bdev, sizeof(float)*N*N);
    hipMalloc(&cdev, sizeof(float)*N*N);

    //a, bの内容をデバイスメモリ上の配列にコピーする
    hipMemcpy(adev, a, sizeof(float)*N*N, hipMemcpyHostToDevice);
    hipMemcpy(bdev, b, sizeof(float)*N*N, hipMemcpyHostToDevice);

    //スレッドブロック数とブロック内スレッド数の設定
    dim3 bdim(BSX, BSY); //ブロック内スレッド数
    dim3 gdim(N/BSX, N/BSY); //スレッドブロック数

    //合計N*N個のスレッドになる．
    //カーネル関数の並列呼び出し
    printf("\nGPU上の行列積を開始します\n");
    mulMatrix<<<gdim, bdim>>>(adev, bdev, cdev);
    hipDeviceSynchronize();

    //計算結果(cdevが指す領域)をホスト側にコピーする
    hipMemcpy(c, cdev, sizeof(float)*N*N, hipMemcpyDeviceToHost);
    printMatrix(c, "c");

    //デバイス上に確保した配列領域を解放する
    hipFree(adev);
    hipFree(bdev);
    hipFree(cdev);

    ////// ここまでGPU版行列積 ////////////////////////////////////////

    ////// ここから， 確認のためのCPU版行列積 ///////////////////////
    printf("\nCPU上の行列積を開始します\n");
    for (int i=0; i<N; i++){
        for (int j=0; j<N; j++){
            cs[i][j] = 0.0;
        }
    }
    for (int i=0; i<N; i++){
        for (int k=0; k<N; k++){
            for (int j=0; j<N; j++){
                cs[i][j] += a[i][k] * b[k][j];
            }
        }
    }

    printMatrix(cs, "cs");

    ////// ここまでCPU版行列積 ////////////////////////////////////////

    return 0;
}